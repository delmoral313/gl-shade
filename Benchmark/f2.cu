#include "hip/hip_runtime.h"
__global__ void F_D(double *Ovector,storage *mem,int *Pvector,double *r25,
double *r50,double *r100,int *s,double *w,row *OvectorVec,ind *pop,int popsize)
{ 
  __shared__ double cache[N_threads];
  double acc,reg;
  int i = blockIdx.x;
  int j = threadIdx.x;

  while (i < popsize) 
  {
    //////////////////////////Process an individual/////////////////////////////////////
    acc = 0.0;
    j = threadIdx.x;
    //Compute rastrigin
    while(j<dim)
    { 
      reg = pop[i].x[j] - Ovector[j];//anotherz[j]
      reg = sign(reg) * exp( hat(reg) + 0.049 * ( sin( c1(reg) * hat(reg) ) + sin( c2(reg)* hat(reg) )  ) ) ;//transform_osz
      if(reg>0) reg = pow(reg, 1 + 0.2 * j/((double) (dim-1)) * sqrt(reg));//transform_asy
      reg = reg * pow(10, 0.5 * j/((double) (dim-1)) );//lambda
      acc += ( reg * reg - 10.0 * cos(2 * PI * reg) + 10.0 );//accumulate
      j += blockDim.x;
    }
    cache[threadIdx.x] = acc;
    __syncthreads();

    // reduction step
    // At the end: cache[0] = cache[0]+cache[1]+cache[2]+.....+cache[NT]
    reduce(cache);//[NT stands for Number of threads]
    ////////////////////////////////////////////////////////////////////
    if(threadIdx.x==0) pop[i].fx = cache[0];
    __syncthreads();
    i += gridDim.x;
  }
}

//OMP
double F_H(double *x)
{
  int j;
  double sum;
  double reg;

  //rastrigin
  sum = 0.0;
  #pragma omp parallel for reduction(+:sum) private(reg) num_threads(maxThreads)
  for (j=0; j<dim; ++j)
  {
    reg = x[j] - Ovector[j];
    reg = sign_h(reg) * exp( hat_h(reg) + 0.049 * ( sin( c1_h(reg) * hat_h(reg) ) + sin( c2_h(reg)* hat_h(reg) )  ) ) ;//transform_osz
    if(reg>0) reg = pow(reg, 1 + 0.2 * j/((double) (dim-1)) * sqrt(reg));//transform_asy
    reg = reg * pow(10, 0.5 * j/((double) (dim-1)) );//lambda
    reg = ( reg * reg - 10.0 * cos(2 * PI * reg) + 10.0 );
    sum += reg;
  }
  return sum;
}


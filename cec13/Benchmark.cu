#include "hip/hip_runtime.h"
//Basic and reading functions.
/**************************READING***************************************/
double* readOvector(int dimension, ushort ID)
{
  // read O vector from file in csv format
  double* d = new double[dimension];
  stringstream ss;
  ss<< "cdatafiles/" << "F" << ID << "-xopt.txt";
  ifstream file (ss.str());
  string value;
  string line;
  int c=0;
  
  if (file.is_open())
    {
      stringstream iss;
      while ( getline(file, line) )
        {
          iss<<line;
          while (getline(iss, value, ','))
            {
              d[c++] = stod(value);
            }
          iss.clear();
        }
      file.close();
    }
  else
    {
      cout<<"Cannot open datafiles"<<endl;
    }
  return d;
}
////////////////////////////////////////////////////////////////////////////////
void readPermVector(int *d,int dimension, ushort ID){

  //int* d;
  //d = new int[dimension];

  stringstream ss;
  ss<< "cdatafiles/" << "F" << ID << "-p.txt";
  ifstream file (ss.str());
  int c=0;
  string value;

  if (file.is_open())
    {
      while (getline(file,value,','))
        {
          d[c++] = stod(value) - 1;
        }
    }
  //return(d);
}
////////////////////////////////////////////////////////////////////////////////
void readS(int *s0,int num, ushort ID)
{
  //int *s0 = new int[num];

  stringstream ss;
  ss<< "cdatafiles/" << "F" << ID << "-s.txt";
  ifstream file (ss.str());
  int c=0;
  string value;
  if (file.is_open())
    {
      while (getline(file,value))
        {
          // cout<<stod(value)<<endl;
          s0[c++] = stod(value);
        }
    }
  //return s0;
}
////////////////////////////////////////////////////////////////////////////////
void readR(double *m,int sub_dim, ushort ID)
{
  //double* m;
  //m = new double[sub_dim*sub_dim];
  // for (int i = 0; i< sub_dim; i++)
  //   {
  //     m[i] = new double[sub_dim];
  //   }

  stringstream ss;
  ss<< "cdatafiles/" << "F" << ID << "-R"<<sub_dim<<".txt";
  // cout<<ss.str()<<endl;

  ifstream file (ss.str());
  string value;
  string line;
  int i=0;
  int j;

  if (file.is_open())
    {
      stringstream iss;
      while ( getline(file, line) )
        {
          j=0;
          iss<<line;
          while (getline(iss, value, ','))
            {
              // printf("%d,%d\t%f\n", i,j, stod(value));
              m[i*sub_dim + j] = stod(value);
              // printf("done\n");
              j++;
            }
          iss.clear();
          i++;
        }
      file.close();
    }
  else
    {
      cout<<"Cannot open datafiles"<<endl;
    }
  //return m;
}
////////////////////////////////////////////////////////////////////////////////
void readW(double *w0,int num, ushort ID)
{
  //double *w0 = new double[num];

  stringstream ss;
  ss<< "cdatafiles/" << "F" << ID << "-w.txt";
  ifstream file (ss.str());
  int c=0;
  string value;
  if (file.is_open())
    {
      while (getline(file,value))
        {
          // cout<<stod(value)<<endl;
          w0[c++] = stod(value);
        }
    }

  //return w0;
}
////////////////////////////////////////////////////////////////////////////////
double** readOvectorVec(ushort ID)
{
  // read O vector from file in csv format, seperated by s_size groups
  double** d = (double**) malloc(s_size*sizeof(double*));
  stringstream ss;
  ss<< "cdatafiles/" << "F" << ID << "-xopt.txt";
  ifstream file (ss.str());
  string value;
  string line;
  int c = 0;                      // index over 1 to dim
  int i = -1;                      // index over 1 to s_size
  int up = 0;                   // current upper bound for one group
  
  if (file.is_open())
    {
      stringstream iss;
      while ( getline(file, line) )
        {
          if (c==up)             // out (start) of one group
            {
              // printf("=\n");
              i++;
              d[i] =  (double*) malloc(s[i]*sizeof(double));
              up += s[i];
            }
          iss<<line;
          while (getline(iss, value, ','))
            {
              // printf("c=%d\ts=%d\ti=%d\tup=%d\tindex=%d\n",c,s[i],i,up,c-(up-s[i]));
              d[i][c-(up-s[i])] = stod(value);
              // printf("1\n");
              c++;
            }
          iss.clear();
          // printf("2\n");
        }
      file.close();
    }
  else
    {
      cout<<"Cannot open datafiles"<<endl;
    }
  return d;  
}
/******************************************************************************/
/////////////////////////////////////////////////////////////////////////
/////////////////////////////C///////////////////////////////////////////
/////////////////////////////U///////////////////////////////////////////
/////////////////////////////D///////////////////////////////////////////
/////////////////////////////A///////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////
/*****************************CUDA*******************************************/
__device__ int sign(double x)
{
  if (x > 0) return 1;
  if (x < 0) return -1;
  return 0;
} 
////////////////////////////////////////////////////////////////////////////////
__device__ double hat(double x)
{
  if (x==0) //if (fabs(x)<= 1e-6)
    {
      return 0;
    }
  else
    {
      return log(abs(x));
    }
}
////////////////////////////////////////////////////////////////////////////////
__device__ double c1(double x)
{
  if (x>0)
    {
      return 10;
    }
  else
    {
      return 5.5;
    }
}
////////////////////////////////////////////////////////////////////////////////
__device__ double c2(double x)
{
  if (x>0)
    {
      return 7.9;
    }
  else
    {
      return 3.1;
    }
}
////////////////////////////////////////////////////////////////////////////////
__device__ void multiplyR25_normal(double *vector,double *matrix,int size,int j,double *y)
{
  /*
  Here it's performed a normal matrix multiplication (No pagination). Pagination means 
  downloading data pages from global memory to shared memory. 
  */
  int l;
  //For every matrix row do...
  while(j<size) //let in all threads which private j variable is less than number of rows 
  { // let thread handle a row 
    y[j] = 0.0;
    for(l = size-1; l >=0; l--) 
      y[j] += (vector[l] * matrix[j*size + l]);
    j += blockDim.x; //advance j: this line is equivalent to say j += number of threads in block
  }
  __syncthreads();
}
////////////////////////////////////////////////////////////////////////////////
__device__ void multiplyR50_pagination(double *vector,double *matrix,int size,int j,
double *y,double *cache,int pagesize,int &mempointer,int &rowcount)
{
  /* multiply_s
  Pagination means that a data page (matrix rows) is loaded from global memory to cache as matrix R50 is too heavy to load it completely.
  The pagination were found useful to speed up computation as reading directly matrix entries from global memory is very slow.
  */
  int n,step;

  //Let thread #0 to set shared variables  
  if (threadIdx.x==0) {rowcount = 0; mempointer = 0;}
  /*If pagesize is set to 1000, then we get step = 20 = 1000/50 meaning that we can load 20 rows in one page */ 
  step = pagesize/size;
  __syncthreads(); //wait for all 

  while(rowcount<40)
  { 
    //Loading page from global memory to cache
    j = threadIdx.x;
    while(j<pagesize)
    {
      cache[j] = matrix[mempointer+j];
      j+=blockDim.x;
    }
    __syncthreads();

    //Processing page or every row 
    if (threadIdx.x<step)
    {
      j = rowcount + threadIdx.x;
      y[j] = 0.0;
      for(n=size-1; n>=0; n--) y[j] += (vector[n] * cache[threadIdx.x*size + n]); 
    }
    if (threadIdx.x==step){rowcount += step; mempointer += pagesize;}
    __syncthreads();
  }

  //Loading last page from global memory to cache
  j = threadIdx.x;
  while(j<500)
  {
    cache[j] = matrix[2000+j];
    j+=blockDim.x;
  }
  __syncthreads();
  //Processing page
  if (threadIdx.x<10)
  {
    j = 40 + threadIdx.x;
    y[j] = 0.0;
    for(n=size-1; n>=0; n--) y[j] += (vector[n] * cache[threadIdx.x*size + n]); 
  }
}
////////////////////////////////////////////////////////////////////////////////
__device__ void multiplyR100_pagination(double *vector,double *matrix,int size,int j,
double *y,double *cache,int pagesize,int &mempointer,int &rowcount)
{
  /* multiply_s
  Pagination means that a data page (matrix rows) is loaded from global memory to cache as matrix R100 is too heavy to load it completely.
  The pagination were found useful to speed up computation as reading directly matrix entries from global memory is very slow.
  */
  int n,step;

  if (threadIdx.x==0) {rowcount = 0; mempointer = 0;}
  /*If pagesize is set to 1500, then we get step = 15 = 1500/100 meaning that we can load 15 rows in one page */ 
  step = pagesize/size;
  __syncthreads();

  while(rowcount<90)
  { 
    //Loading page from global memory to cache
    j = threadIdx.x;
    while(j<pagesize)
    {
      cache[j] = matrix[mempointer+j];
      j+=blockDim.x;
    }
    __syncthreads();

    //Processing page
    if (threadIdx.x<step)
    {
      j = rowcount + threadIdx.x;
      y[j] = 0.0;
      for(n=size-1; n>=0; n--) y[j] += (vector[n] * cache[threadIdx.x*size + n]); 
    }
    if (threadIdx.x==step){rowcount += step; mempointer += pagesize;}
    __syncthreads();
  }

  //Loading last page from global memory to cache
  j = threadIdx.x;
  while(j<1000)
  {
    cache[j] = matrix[9000+j];
    j+=blockDim.x;
  }
  __syncthreads();
  //Processing page
  if (threadIdx.x<10)
  {
    j = 90 + threadIdx.x;
    y[j] = 0.0;
    for(n=size-1; n>=0; n--) y[j] += (vector[n] * cache[threadIdx.x*size + n]); 
  }
}
////////////////////////////////////////////////////////////////////////////////
__device__ void reduce(double *cache)
{
  //Perform a dichotomous reduction:cache[0] = cache[0] + cache[1] + ... + cache[number of threads]
  int k = blockDim.x/2;
  while (k != 0) 
  {
    if (threadIdx.x < k) cache[threadIdx.x] += cache[threadIdx.x + k];
    __syncthreads();
    k /= 2; 
  }
  __syncthreads();
}
__device__ void reduce_twice(double *cache,double *cache2)
{
  //Perform a dichotomous reduction: cache[0] = cache[0] + cache[1] + ... + cache[number of threads] 
  int k = blockDim.x/2;
  while (k != 0) 
  {
    if (threadIdx.x < k) 
    {
      cache[threadIdx.x] += cache[threadIdx.x + k];
      cache2[threadIdx.x] += cache2[threadIdx.x + k];
    }
    __syncthreads();
    k /= 2; 
  }
  __syncthreads();
}
/******************************************************************************/
/////////////////////////////////////////////////////////////////////////
/////////////////////////////O///////////////////////////////////////////
/////////////////////////////M///////////////////////////////////////////
/////////////////////////////P///////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////
/************************OMP*********************************************/
int sign_h(double x)
{
  if (x > 0) return 1;
  if (x < 0) return -1;
  return 0;
} 
////////////////////////////////////////////////////////////////////////////////
double hat_h(double x)
{
  if (x==0) //if (fabs(x)<= 1e-6)
    {
      return 0;
    }
  else
    {
      return log(abs(x));
    }
}
////////////////////////////////////////////////////////////////////////////////
double c1_h(double x)
{
  if (x>0)
    {
      return 10;
    }
  else
    {
      return 5.5;
    }
}
////////////////////////////////////////////////////////////////////////////////
double c2_h(double x)
{
  if (x>0)
    {
      return 7.9;
    }
  else
    {
      return 3.1;
    }
}
////////////////////////////////////////////////////////////////////////////////
void multiply_h(double *vector,double *matrix,int size,double *y)
{
  int m,n;

  #pragma omp parallel for num_threads(maxThreads)
  for (m=0; m<size; ++m)
  {
    y[m] = 0.0;
    for (n=0; n<size; ++n) y[m] += (vector[n] * matrix[m*size + n]);
  }
}
////////////////////////////////////////////////////////////////////////////////


#include "hip/hip_runtime.h"
/*
Tanabe, R.; Fukunaga, A., "Success-history based parameter adaptation
for Differential Evolution," Evolutionary Computation (CEC), 2013 IEEE
Congress on , vol., no., pp.71,78, 20-23 June 2013
doi:10.1109/CEC.2013.655755510.1109/CEC.2013.6557555

Wan-li Xiang, Xue-lei Meng, Mei-qing An, Yinzhen Li, and Ming-xia Gao.
An enhanced differential evolution algorithm based on multiple mutation 
strategies. 
Computational Intelligence and Neuroscience, 2015:1–15, 11 2015.
*/
/******************************************************************************/
/////////////////////////////eSHADE_ls//////////////////////////////////
/******************************************************************************/
class eshade_ls
{
private:
	//host
	int NP,D,H_maxsize,stop_criterion,k,counter;
	float lb,ub,p_min,p_max;
	vector<ind> A;
	vector<ind> A_tmp;
	vector<ind> memory;
	evol_data_struct2 *evol_data;
	rank_ind *ranklist;
	ind *child;
	double *S_F;
	double *S_Cr;
	double *W;
	int S_size;
	double uF,uCr;
	double mean[2];
	double *mu;

	//device
	ind *pop_D;
	evol_data_struct2 *evol_data_D;
	rank_ind *rank_D;
	ind *memory_D;
	ind *child_D;
	double *S_F_D;
	double *S_Cr_D;
	double *W_D;
	double *mean_D;
	ind *child_mu_D;

	void update_best()
	{
		int i;
		double min = pop[0].fx; int min_id = 0;

		for (i = 1; i < NP; ++i)
		{
			if (pop[i].fx < min)
	    	{
	      		min = pop[i].fx;
	      		min_id = i;
	    	}

	  	}
	  	best = min_id;
	}

	void apply_A_maintenance()
	{	
		int r;
		while (A.size() > NP)
		{
			r = rnd(0,A.size()-1);
			A.erase(A.begin()+r);
		}
	}

	void sort(rank_ind *S)
	{	/*
		Insertion sort
		*/

		int l,m;
		rank_ind key;

		for (l = 1; l < NP; ++l)
		{
			key = S[l];
			//Insert S[l] 􏱾 into the sorted sequence S[1......l-1]
			m = l-1;
			while(m>=0 && S[m].fitness>key.fitness)
			{
				S[m+1] = S[m];
				m--;
			}
			S[m+1] = key;
		}
	}

	void ls_search(int maxFEs_LS_in_round,ind &global_best,int &glshade_current_FEs,int glshade_stop_criterion)
	{
		int totalevals = 0;
	    double score; // trial vector fitness
	    float wmax = 0.2; float wmin = 0; float r2; int k0,l,n,j;

		while (totalevals<maxFEs_LS_in_round && glshade_current_FEs<glshade_stop_criterion)
		{
		    for (j = 0; j < D; ++j)
		    { 
		      	if (glshade_current_FEs>=glshade_stop_criterion || totalevals>=maxFEs_LS_in_round) break;

		        // set mu = x_best
		        for (l = 0; l < D; ++l) mu[l] = pop[best].x[l];

		        do k0 = rnd(0,NP-1); while(k0==best);
		        do n = rnd(0,D-1); while(n==j);

		        // compute r2
		        r2 = wmin + ((glshade_current_FEs/glshade_stop_criterion)*(wmax-wmin));

		        // perturb
		        if (flip(r2))
		            mu[j] = pop[best].x[n] + ((2*rndreal(0,1))-1)*(pop[best].x[n] - pop[k0].x[n]);
		        else
		            mu[j] = pop[best].x[j] + ((2*rndreal(0,1))-1)*(pop[best].x[n] - pop[k0].x[n]);

		        // making sure a gen isn't out of boundary
		        if (mu[j] > ub)
		            mu[j] = (ub+pop[best].x[j])/2;
		        else if (mu[j] < lb)
		            mu[j] = (lb+pop[best].x[j])/2;

		        // evaluate mu
		        score = F_H(mu); // F_H <- OMP implementation 
		        glshade_current_FEs += 1; totalevals += 1; counter += 1;

		        // choose better{mu,current_best} as new best
		        if (score <= pop[best].fx)
		        {
			      	if (glshade_current_FEs<=glshade_stop_criterion && score<global_best.fx)
			      	{
		        		global_best.FEs_when_found = glshade_current_FEs;
		        		global_best.fx = score;
		        		memcpy(global_best.x,mu,D*sizeof(double));
			      	}
		            for (l = 0; l < D; ++l) pop[best].x[l] = mu[l];
		            pop[best].fx = score;	        	
		        }
				if(glshade_current_FEs==1.2e5 || glshade_current_FEs==3e5 || glshade_current_FEs==6e5 || 
				glshade_current_FEs==9e5 || glshade_current_FEs==1.2e6 || glshade_current_FEs==1.5e6
				|| glshade_current_FEs==1.8e6 || glshade_current_FEs==2.1e6 || glshade_current_FEs==2.4e6 || 
				glshade_current_FEs==2.7e6 || glshade_current_FEs==3e6)
					fprintf(file_results,"%d,%d,%.2f,%.6e\n",glshade_current_FEs,ID,Rseed,global_best.fx);
		    }
		}
	  return;
	}

public:
	int best; //position where the best individual is
	ind *pop; //ind data type is defined in in objective_function.h
	double *M_Cr;
	double *M_F;

	eshade_ls(float lowbound, float upbound, int popsize=100, int memory_size=100, int N_decison_var=1000,int eshadels_stop_criterion=25000) 
	{	
		//__init__
		/* lb and ub are needed while executing eshade_ls on host since
		the perturbation EDE-LS method is done on host*/
		lb = lowbound;
		ub = upbound;
		NP = popsize; 
		D = N_decison_var;
		stop_criterion = eshadels_stop_criterion;
		H_maxsize = memory_size;
		k = 0;
		p_min = 2.0/NP;
		p_max = 0.1;
		pop = (ind *)malloc(NP*sizeof(ind));
		ranklist = (rank_ind *)malloc(NP*sizeof(rank_ind));
		evol_data = (evol_data_struct2 *)malloc(NP*sizeof(evol_data_struct2));
		child = (ind *)malloc(NP*sizeof(ind));
		S_F = (double *)malloc(NP*sizeof(double));
		S_Cr = (double *)malloc(NP*sizeof(double));
		W = (double *)malloc(NP*sizeof(double));
		M_F = (double *)malloc(H_maxsize*sizeof(double));
		M_Cr = (double *)malloc(H_maxsize*sizeof(double));
		mu = (double *)malloc(D*sizeof(double));
	}

	void init_population_in_device(hiprandState *state_D,ind &global_best,int &glshade_current_FEs)
	{
		int i;

		// 1. allocate memory 
		hipMalloc(&pop_D,NP*sizeof(ind));

		//3. Initializing population on device while updating current evaluations on host 
		init_population<<<N_blocks,N_threads>>>(state_D,pop_D,NP);
		glshade_current_FEs += NP;

		//4. Evaluating the created population on device while initializing M_F and M_Cr on host 
		F_D<<<N_blocks,N_threads>>>(Ovector_D,mem_D,Pvector_D,r25_D,r50_D,r100_D,s_D,w_D,OvectorVec_D,pop_D,NP);
		for (i = 0; i < H_maxsize; ++i) {M_F[i] = 0.5; M_Cr[i] = 0.5;} // Init Cr and F storage 

		//5. copy data from device to host
		hipMemcpy(pop,pop_D,NP*sizeof(ind),hipMemcpyDefault);

		//6. free memory
		hipFree(pop_D);

		//7. update best and record it 
		update_best();
		if (pop[best].fx < global_best.fx)
			global_best = pop[best]; //set the best individual of population 2 as global best
	}

	void evolve_in_device(default_random_engine &rng,hiprandState *state_D,ind &global_best,int &glshade_current_FEs,int glshade_stop_criterion)
	{

		//Integrate global_best to population (receive)
		int r;
		do r = rnd(0,NP-1); while(r==best); //choose a random position to place it
		pop[r] = global_best; //place it at position r

		//Set counter and storage size counter
		counter = 0;
		S_size = 0;
		int L,i,j;

	    //Allocate memory on device
	    hipMalloc(&evol_data_D,NP*sizeof(evol_data_struct2));
	    hipMalloc(&pop_D,NP*sizeof(ind));
	    hipMalloc(&child_mu_D,1*sizeof(ind));
	    hipMalloc(&rank_D,NP*sizeof(rank_ind));
	    hipMalloc(&memory_D,2*NP*sizeof(ind));
	    hipMalloc(&child_D,NP*sizeof(ind));
	    hipMalloc(&S_F_D,NP*sizeof(double));
	    hipMalloc(&S_Cr_D,NP*sizeof(double));
	    hipMalloc(&W_D,NP*sizeof(double));
	    hipMalloc(&mean_D,2*sizeof(double));

	    //While stopping condition is not met:
		while(counter<stop_criterion && glshade_current_FEs<glshade_stop_criterion)
		{
			/******************* SHADE ************************/
			// Join Population and external archive
			memory.insert(memory.end(), &pop[0], &pop[NP]); //memory = pop;
			memory.insert(memory.end(), A.begin(), A.end());

	    	//Prepare random data
		    for (i = 0; i < NP; ++i)
		    {
		        ranklist[i].id = i; ranklist[i].fitness = pop[i].fx;

				/*******************Setting F and Cr************************/
				// Generate F and Cr using a normal distribution with mean
				// taken randomly.
				r = rnd(0,H_maxsize-1);
				uF = M_F[r]; normal_distribution<double> Ndistribution_F(uF,0.1);
				uCr = M_Cr[r]; normal_distribution<double> Ndistribution_Cr(uCr,0.1);

		        evol_data[i].Cr = Ndistribution_Cr(rng); 
		        if (evol_data[i].Cr > 1.0) evol_data[i].Cr = 1.0; 
		        else if(evol_data[i].Cr < 0.0) evol_data[i].Cr = 0.0;

		        evol_data[i].F = Ndistribution_F(rng); 
		        if (evol_data[i].F > 1.0) evol_data[i].F = 1.0; 
		        while (evol_data[i].F <= 0.0) evol_data[i].F = Ndistribution_F(rng);

		        /*******************Setting p_best************************/
		        evol_data[i].p_best = rnd(0,int(rndreal(p_min,p_max)*NP)); // take an index within best range

		        /*******************Choosing a and b************************/
		        // randomly pick 2 different members
		        do evol_data[i].a = rnd(0,NP-1); while(evol_data[i].a==i); // from pop
		        do evol_data[i].b = rnd(0,memory.size()-1); while(evol_data[i].b==i || evol_data[i].b==evol_data[i].a); // from pop U archive

		        /*******************Get exp crossover window************************/
		        evol_data[i].Jrand = j = rnd(0,D-1);
		        L = 0;
		        do {evol_data[i].Jend = j; j = (j+1)%D; L++;} while(flip(evol_data[i].Cr) and L<D);
		    }
		    //Rank population by fitness
		    sort(ranklist);//sort by fitness min => ranklist[0].fitness

			//Load generated data and current population to device 
			hipMemcpy(evol_data_D,evol_data,NP*sizeof(evol_data_struct2),hipMemcpyDefault);
			hipMemcpy(pop_D,pop,NP*sizeof(ind),hipMemcpyDefault);
			hipMemcpy(rank_D,ranklist,NP*sizeof(rank_ind),hipMemcpyDefault);
			hipMemcpy(memory_D,memory.data(),memory.size()*sizeof(ind),hipMemcpyDefault);

			//Lauch kernel: mutation,recombination and function evaluation 
			eshade_ls_engine<<<N_blocks,N_threads>>>(evol_data_D,pop_D,rank_D,memory_D,child_D,NP);
			F_D<<<N_blocks,N_threads>>>(Ovector_D,mem_D,Pvector_D,r25_D,r50_D,r100_D,s_D,w_D,OvectorVec_D,child_D,NP);
			hipMemcpy(child,child_D,NP*sizeof(ind),hipMemcpyDefault);

			//Selection
			for (i = 0; i < NP; ++i)
			{
				//Update FEs counter 
				glshade_current_FEs += 1; counter += 1;
		        if (child[i].fx <= pop[i].fx) // if better than target vector then:
		        {	
		        	//if strictly better then:
		        	if (child[i].fx < pop[i].fx)
		        	{
		        		A_tmp.push_back(pop[i]);//add defeated parent to external archive
		        		S_F[S_size] = evol_data[i].F;//record F
		        		S_Cr[S_size] = evol_data[i].Cr;//record Cr
		        		W[S_size] = pop[i].fx - child[i].fx;//record improvement
		        		S_size++;//increase storage size counter
		        	}

		        	//update global_best if needed 
		        	if (child[i].fx<global_best.fx && glshade_current_FEs<=glshade_stop_criterion)
		        	{
		        		global_best = child[i];
		        		global_best.FEs_when_found = glshade_current_FEs;
		        	}
		        	//Advance child to next generation
		        	pop[i] = child[i];
		        }
				if(glshade_current_FEs==1.2e5 || glshade_current_FEs==3e5 || glshade_current_FEs==6e5 || 
				glshade_current_FEs==9e5 || glshade_current_FEs==1.2e6 || glshade_current_FEs==1.5e6
				|| glshade_current_FEs==1.8e6 || glshade_current_FEs==2.1e6 || glshade_current_FEs==2.4e6 || 
				glshade_current_FEs==2.7e6 || glshade_current_FEs==3e6)
					fprintf(file_results,"%d,%d,%.2f,%.6e\n",glshade_current_FEs,ID,Rseed,global_best.fx);
			}
			//If F and Cr storages are non-empty
			if (S_size > 0)
			{	//Load F, Cr and W data to device  
				hipMemcpy(S_F_D,S_F,S_size*sizeof(double),hipMemcpyDefault);
				hipMemcpy(S_Cr_D,S_Cr,S_size*sizeof(double),hipMemcpyDefault);
				hipMemcpy(W_D,W,S_size*sizeof(double),hipMemcpyDefault);
				mean_WAWL<<<2,64>>>(S_Cr_D,S_F_D,W_D,S_size,mean_D); //Compute mean WA and mean WL in device
			}

			//Concurrently update best solution index
			update_best();

			// Concurrently check external archive
			A.insert(A.end(), A_tmp.begin(), A_tmp.end()); // add defeated parents to A
			apply_A_maintenance();//|A| must be less than or equal to popsize

			// Update M_CR and M_F
			if (S_size > 0)
			{	//Record means 
				hipMemcpy(mean,mean_D,2*sizeof(double),hipMemcpyDefault);
				M_F[k] = mean[1]; //weighted Lehmer mean (WL)
				M_Cr[k] = mean[0]; //weighted arithmetic mean (WA)
				k = (k + 1) % H_maxsize;
			}

			// Reset and go again
			S_size = 0;
			A_tmp.clear();
			memory.clear();

			//Apply EDE_LS
			ls_search(D,global_best,glshade_current_FEs,glshade_stop_criterion);
		}
	    //Free memory
	    hipFree(evol_data_D);
	    hipFree(pop_D);
	    hipFree(child_mu_D);
	    hipFree(rank_D);
	    hipFree(memory_D);
	    hipFree(child_D);
	    hipFree(S_F_D);
	    hipFree(S_Cr_D);
	    hipFree(W_D);
	    hipFree(mean_D);

	}

	void free_memory()
	{
		free(pop);
		free(ranklist);
		free(evol_data);
		free(child);
		free(S_F);
		free(S_Cr);
		free(W);
		free(M_F);
		free(M_Cr);
		free(mu);
	}
};


#include "hip/hip_runtime.h"
/*
Tanabe, R.; Fukunaga, A., "Success-history based parameter adaptation
for Differential Evolution," Evolutionary Computation (CEC), 2013 IEEE
Congress on , vol., no., pp.71,78, 20-23 June 2013
doi:10.1109/CEC.2013.655755510.1109/CEC.2013.6557555
*/

/***************** shade class *********************/
class shade
{	// Private stuff
	//Host data//
	int H_maxsize;
	int k;
	int counter;
	float p_min;
	float p_max;
	double *M_Cr;
	double *M_F;
	vector<ind> A;
	vector<ind> A_tmp;
	vector<ind> memory;
	evol_data_struct *evol_data;
	rank_ind *ranklist;
    ind *child;
	double *S_F;
	double *S_Cr;
	double *W;
	int S_size;
	double uF,uCr;
	int r;
    double mean[2];
    //Device data//
    evol_data_struct *evol_data_D;
	rank_ind *rank_D;
	ind *memory_D;
	ind *child_D;
	double *S_F_D;
	double *S_Cr_D;
	double *W_D;
	double *mean_D; 
public:
	ind *pop;
	ind *pop_D;
	int i,j;
	int best_location;
	float lb,ub;
	int NP,D;
	unsigned maxFEs,maxFEs_per_round;
	void set_parameters(int,int,unsigned,unsigned,float,float);
	void init_population_in_device(hiprandState*,unsigned&,ind&);
	void update_best();
	void free_memory();
	void evolve_in_device(unsigned&,default_random_engine&,ind&,hiprandState*);
	void apply_A_maintenance();
	double mean_WA();
	double mean_WL();
	void receive(ind&);
	void sort(rank_ind*);
};

void shade::set_parameters(int popsize,int dimension,unsigned maxEvals,unsigned FEs_per_application,
float lowerbound,float upperbound)
{	
	D = dimension;
	NP = popsize;
	maxFEs = maxEvals;
	maxFEs_per_round = FEs_per_application;
	lb = lowerbound;
	ub = upperbound;
	H_maxsize = 100;//NP
	best_location = 0;
	k = 0;
	counter = 0;
	p_min = 2.0/NP;
	p_max = 0.2; 
	pop = (ind *)malloc(NP*sizeof(ind));
	ranklist = (rank_ind *)malloc(NP*sizeof(rank_ind));
	evol_data = (evol_data_struct *)malloc(NP*sizeof(evol_data_struct));
	child = (ind *)malloc(NP*sizeof(ind));
	S_F = (double *)malloc(NP*sizeof(double));
	S_Cr = (double *)malloc(NP*sizeof(double));
	W = (double *)malloc(NP*sizeof(double));
	M_F = (double *)malloc(H_maxsize*sizeof(double));
	M_Cr = (double *)malloc(H_maxsize*sizeof(double));
}

void shade::init_population_in_device(hiprandState *state_D,unsigned &current_FEs,ind &current_best)
{
	// 1. allocate memory 
	hipMalloc(&pop_D,NP*sizeof(ind));

	// 3. Running kernel
	init_population<<<N_blocks,N_threads>>>(state_D,pop_D,current_FEs,NP);
	current_FEs += NP;
	F_D<<<N_blocks,N_threads>>>(Ovector_D,mem_D,Pvector_D,r25_D,r50_D,r100_D,s_D,w_D,OvectorVec_D,pop_D,NP);

  	// Init Cr and F storage 
	for (i = 0; i < H_maxsize; ++i) {M_F[i] = 0.5; M_Cr[i] = 0.5;}

	//4. copy data from device to host
	hipMemcpy(pop,pop_D,NP*sizeof(ind),hipMemcpyDefault);

	//5. free memory
	hipFree(pop_D);

	//Update best_location and record it
	update_best();
	current_best = pop[best_location];
}

void shade::evolve_in_device(unsigned &current_FEs,default_random_engine &rng,
ind &global_best,hiprandState *state_D)
{
	//Set counter and storage size counter
	counter = 0;
	S_size = 0;

    //Allocate memory on device
    hipMalloc(&evol_data_D,NP*sizeof(evol_data_struct));
    hipMalloc(&pop_D,NP*sizeof(ind));
    hipMalloc(&rank_D,NP*sizeof(rank_ind));
    hipMalloc(&memory_D,2*NP*sizeof(ind));
    hipMalloc(&child_D,NP*sizeof(ind));
    hipMalloc(&S_F_D,NP*sizeof(double));
    hipMalloc(&S_Cr_D,NP*sizeof(double));
    hipMalloc(&W_D,NP*sizeof(double));
    hipMalloc(&mean_D,2*sizeof(double));

    //While stopping condition is not met:
	while(counter<maxFEs_per_round && current_FEs<maxFEs)
	{
		/******************* SHADE ************************/
		// Join Population and external archive
		memory.insert(memory.end(), &pop[0], &pop[NP]); //memory = pop;
		memory.insert(memory.end(), A.begin(), A.end());

    	// Rank population by fitness and prepare random data
	    for (i = 0; i < NP; ++i)
	    {
	        ranklist[i].id = i; ranklist[i].fitness = pop[i].fx;

			/*******************Setting F and Cr************************/
			// Generate F and Cr using a normal distribution with mean
			// taken randomly from storage and std. 0.1
			r = rnd(0,H_maxsize-1);
			uF = M_F[r]; normal_distribution<double> Ndistribution_F(uF,0.1);
			uCr = M_Cr[r]; normal_distribution<double> Ndistribution_Cr(uCr,0.1);

	        evol_data[i].Cr = Ndistribution_Cr(rng); 
	        if (evol_data[i].Cr > 1.0) evol_data[i].Cr = 1.0; 
	        else if(evol_data[i].Cr < 0.0) evol_data[i].Cr = 0.0;

	        evol_data[i].F = Ndistribution_F(rng); 
	        if (evol_data[i].F > 1.0) evol_data[i].F = 1.0; 
	        while (evol_data[i].F <= 0.0) evol_data[i].F = Ndistribution_F(rng);

	        /*******************Setting p_best************************/
	        evol_data[i].p_best = rnd(0,int(rndreal(p_min,0.2)*NP)); // take an index within best pop range

	        /*******************Choosing a and b************************/
	        // randomly pick 2 different members
	        do evol_data[i].a = rnd(0,NP-1); while(evol_data[i].a==i); // from pop
	        do evol_data[i].b = rnd(0,memory.size()-1); while(evol_data[i].b==i || evol_data[i].b==evol_data[i].a); // from pop U archive

	        /*******************Get j_rand************************/
	        evol_data[i].j_rand = rnd(0,D-1);
	    }
	    sort(ranklist);//sort by fitness min => ranklist[0].fitness 

		//Load generated data and current population to device 
		hipMemcpy(evol_data_D,evol_data,NP*sizeof(evol_data_struct),hipMemcpyDefault);
		hipMemcpy(pop_D,pop,NP*sizeof(ind),hipMemcpyDefault);
		hipMemcpy(rank_D,ranklist,NP*sizeof(rank_ind),hipMemcpyDefault);
		hipMemcpy(memory_D,memory.data(),memory.size()*sizeof(ind),hipMemcpyDefault);

		//Lauch kernel: mutation,recombination and function evaluation 
		shade_engine<<<N_blocks,N_threads>>>(state_D,evol_data_D,pop_D,rank_D,memory_D,child_D,NP);
		F_D<<<N_blocks,N_threads>>>(Ovector_D,mem_D,Pvector_D,r25_D,r50_D,r100_D,s_D,w_D,OvectorVec_D,child_D,NP);
		hipMemcpy(child,child_D,NP*sizeof(ind),hipMemcpyDefault);
		
		//Selection
		for (i = 0; i < NP; ++i)
		{
			//Update FEs counter 
			current_FEs += 1; counter += 1;
	        if (child[i].fx <= pop[i].fx) // if better than target vector then:
	        {	
	        	//if strictly better then:
	        	if (child[i].fx < pop[i].fx)
	        	{
	        		A_tmp.push_back(pop[i]);//add defeated parent to external archive
	        		S_F[S_size] = evol_data[i].F;//record F
	        		S_Cr[S_size] = evol_data[i].Cr;//record Cr
	        		W[S_size] = pop[i].fx - child[i].fx;//record improvement
	        		S_size++;//increase storage size counter
	        	}

	        	//update global_best if needed 
	        	if (child[i].fx<global_best.fx && current_FEs<=maxFEs)
	        	{
	        		global_best = child[i];
	        		global_best.FEs_when_found = current_FEs;
	        	}
	        	//Advance child to next generation
	        	pop[i] = child[i];
	        }
			if(current_FEs==1.2e5 || current_FEs==3e5 || current_FEs==6e5 || current_FEs==9e5 || current_FEs==1.2e6 || current_FEs==1.5e6
			|| current_FEs==1.8e6 || current_FEs==2.1e6 || current_FEs==2.4e6 || current_FEs==2.7e6 || current_FEs==3e6)
				fprintf(file_results,"%u,%d,%.2f,%.6e\n",current_FEs,ID,Rseed,global_best.fx);

		}
		//If F and Cr storages are non-empty
		if (S_size > 0)
		{	//Load F, Cr and W data to device  
			uF = double(S_size); //uF is used as temporay variable
			hipMemcpy(S_F_D,S_F,S_size*sizeof(double),hipMemcpyDefault);
			hipMemcpy(S_Cr_D,S_Cr,S_size*sizeof(double),hipMemcpyDefault);
			hipMemcpy(W_D,W,S_size*sizeof(double),hipMemcpyDefault);
			hipMemcpy(&mean_D[0],&uF,1*sizeof(double),hipMemcpyDefault);
			//Compute mean WA and mean WL in device
			mean_WAWL<<<2,64>>>(S_Cr_D,S_F_D,W_D,mean_D);
		}

		// Concurrently update best solution index
		update_best();

		// Concurrently check external archive
		A.insert(A.end(), A_tmp.begin(), A_tmp.end()); // add defeated parents to A
		apply_A_maintenance();//|A| must be less than or equal to popsize

		// Update M_CR and M_F
		if (S_size > 0)
		{	//Record means 
			hipMemcpy(mean,mean_D,2*sizeof(double),hipMemcpyDefault);
			M_F[k] = mean[1];//mean_WL();
			M_Cr[k] = mean[0];//mean_WA();
			k = (k + 1) % H_maxsize;
		}

		// Reset and go again
		S_size = 0;
		A_tmp.clear();
		memory.clear();
	}
    //Free memory
    hipFree(evol_data_D);
    hipFree(pop_D);
    hipFree(rank_D);
    hipFree(memory_D);
    hipFree(child_D);
    hipFree(S_F_D);
    hipFree(S_Cr_D);
    hipFree(W_D);
    hipFree(mean_D);
}

void shade::free_memory()
{
	free(pop);
	free(ranklist);
    free(evol_data);
    free(child);
    free(S_F);
    free(S_Cr);
    free(W);
    free(M_F);
    free(M_Cr);
}

void shade::update_best()
{
  double min = pop[0].fx; int min_id = 0;

  for (i = 1; i < NP; ++i)
  {
    if (pop[i].fx < min)
    {
      min = pop[i].fx;
      min_id = i;
    }

  }
  best_location = min_id;
}

void shade::receive(ind &global_best)
{
	// Place it at best_id index position
	pop[best_location] = global_best;
}

void shade::apply_A_maintenance()
{	
	int r;
	while (A.size() > NP)
	{
		r = rnd(0,A.size()-1);
		A.erase(A.begin()+r);
	}
}


double shade::mean_WA()
{	// W -> improvement
    int k,g;
    int size = S_size;
    double delta_sum = 0;
    double tmp = 0;
    
    for (g = 0; g < size; g++)
        delta_sum += W[g];
    
    for (k = 0; k < size; k++)
        tmp += (W[k]/delta_sum) * S_Cr[k];

    if (tmp > 1) tmp = 1;
    else if (tmp < 0) tmp = 0;
    
    return tmp;
}

double shade::mean_WL()
{	// W -> improvement
    int k,g;
    int size = S_size;
    double delta_sum = 0;
    double tmp1 = 0;
    double tmp2 = 0;
    double res;
    
    for (g = 0; g < size; g++)
        delta_sum += W[g];
    
    for (k = 0; k < size; k++)
        tmp1 += (W[k]/delta_sum) * (S_F[k]*S_F[k]);
    for (k = 0; k < size; k++)
        tmp2 += (W[k]/delta_sum) * S_F[k];
    
    res = tmp1/tmp2;
    if (res > 1) res = 1;
    else if (res < 0) res = 0;

    return res; 
}

//sort(ranklist.begin(),ranklist.end(), [](rank_ind i1, rank_ind i2) {return (i1.fitness < i2.fitness);});
void shade::sort(rank_ind *S)
{	/*Insertion sort*/
	int l,m;
	rank_ind key;

	for (l = 1; l < NP; ++l)
	{
		key = S[l];
		//Insert S[l] 􏱾 into the sorted sequence S[1......l-1]
		m = l-1;
		while(m>=0 && S[m].fitness>key.fitness)
		{
			S[m+1] = S[m];
			m--;
		}
		S[m+1] = key;
	}
}




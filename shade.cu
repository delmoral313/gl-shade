#include "hip/hip_runtime.h"
/*
Tanabe, R.; Fukunaga, A., "Success-history based parameter adaptation
for Differential Evolution," Evolutionary Computation (CEC), 2013 IEEE
Congress on , vol., no., pp.71,78, 20-23 June 2013
doi:10.1109/CEC.2013.655755510.1109/CEC.2013.6557555
*/
/******************************************************************************/
/////////////////////////////SHADE//////////////////////////////////
/******************************************************************************/
class shade
{
private:
	//host
	int NP,D,H_maxsize,stop_criterion,k;
	float p_min,p_max;
	vector<ind> A;
	vector<ind> A_tmp;
	vector<ind> memory;
	evol_data_struct *evol_data;
	rank_ind *ranklist;
	ind *child;
	double *S_F;
	double *S_Cr;
	double *W;
	int S_size;
	double uF,uCr;
	double mean[2];

	//device
	evol_data_struct *evol_data_D;
	rank_ind *rank_D;
	ind *pop_D;
	ind *memory_D;
	ind *child_D;
	double *S_F_D;
	double *S_Cr_D;
	double *W_D;
	double *mean_D; 

	void update_best()
	{
		int i;
		double min = pop[0].fx; int min_id = 0;

		for (i = 1; i < NP; ++i)
		{
			if (pop[i].fx < min)
	    	{
	      		min = pop[i].fx;
	      		min_id = i;
	    	}

	  	}
	  	best = min_id;
	}

	void apply_A_maintenance()
	{	
		int r;
		while (A.size() > NP)
		{
			r = rnd(0,A.size()-1);
			A.erase(A.begin()+r);
		}
	}

	void sort(rank_ind *S)
	{	/*Insertion sort*/
		int l,m;
		rank_ind key;

		for (l = 1; l < NP; ++l)
		{
			key = S[l];
			//Insert S[l] 􏱾 into the sorted sequence S[1......l-1]
			m = l-1;
			while(m>=0 && S[m].fitness>key.fitness)
			{
				S[m+1] = S[m];
				m--;
			}
			S[m+1] = key;
		}
	}

public:
	int best; //position where the best individual is
	ind *pop; //ind data type is defined in in objective_function.h 
	double *M_Cr;
	double *M_F;

	shade(float lb, float ub, int popsize=100, int memory_size=100, int N_decison_var=1000,int shade_stop_criterion=25000)
	{	
		//__init__
		/* lb and ub are no needed while executing shade on host since
		mutation and recombination is done on device */
		NP = popsize; 
		D = N_decison_var;
		stop_criterion = shade_stop_criterion;
		H_maxsize = memory_size;
		k = 0;
		p_min = 2.0/NP;
		p_max = 0.2;
		pop = (ind *)malloc(NP*sizeof(ind));
		ranklist = (rank_ind *)malloc(NP*sizeof(rank_ind));
		evol_data = (evol_data_struct *)malloc(NP*sizeof(evol_data_struct));
		child = (ind *)malloc(NP*sizeof(ind));
		S_F = (double *)malloc(NP*sizeof(double));
		S_Cr = (double *)malloc(NP*sizeof(double));
		W = (double *)malloc(NP*sizeof(double));
		M_F = (double *)malloc(H_maxsize*sizeof(double));
		M_Cr = (double *)malloc(H_maxsize*sizeof(double));
	}

	void init_population_in_device(hiprandState *state_D,ind &global_best,int &glshade_current_FEs)
	{
		int i;

		// 1. allocate memory 
		hipMalloc(&pop_D,NP*sizeof(ind));

		//3. Initializing population on device while updating current evaluations on host 
		init_population<<<N_blocks,N_threads>>>(state_D,pop_D,NP);
		glshade_current_FEs += NP;

		//4. Evaluating the created population on device while initializing M_F and M_Cr on host 
		F_D<<<N_blocks,N_threads>>>(Ovector_D,mem_D,Pvector_D,r25_D,r50_D,r100_D,s_D,w_D,OvectorVec_D,pop_D,NP);
		for (i = 0; i < H_maxsize; ++i) {M_F[i] = 0.5; M_Cr[i] = 0.5;} // Init Cr and F storage 

		//5. copy data from device to host
		hipMemcpy(pop,pop_D,NP*sizeof(ind),hipMemcpyDefault);

		//6. free memory
		hipFree(pop_D);

		//7. update best and record it 
		update_best();
		global_best = pop[best]; //set the best individual of population 1 as global best
	}

	void evolve_in_device(default_random_engine &rng,hiprandState *state_D,ind &global_best,int &glshade_current_FEs,int glshade_stop_criterion)
	{

		//Integrate global_best to population (receive)
		pop[best] = global_best; // place it at best_id index position

		//Set counter, storage size counter 
		int counter = 0;
		int S_size = 0;
		int r,i;

	    //Allocate memory on device
	    hipMalloc(&evol_data_D,NP*sizeof(evol_data_struct));
	    hipMalloc(&pop_D,NP*sizeof(ind));
	    hipMalloc(&rank_D,NP*sizeof(rank_ind));
	    hipMalloc(&memory_D,2*NP*sizeof(ind));
	    hipMalloc(&child_D,NP*sizeof(ind));
	    hipMalloc(&S_F_D,NP*sizeof(double));
	    hipMalloc(&S_Cr_D,NP*sizeof(double));
	    hipMalloc(&W_D,NP*sizeof(double));
	    hipMalloc(&mean_D,2*sizeof(double));

	    //While stopping condition is not met:
		while(counter<stop_criterion && glshade_current_FEs<glshade_stop_criterion)
		{
			/******************* SHADE ************************/
			// Join Population and external archive
			memory.insert(memory.end(), &pop[0], &pop[NP]); //memory = pop;
			memory.insert(memory.end(), A.begin(), A.end()); 

	    	//Prepare random data
		    for (i = 0; i < NP; ++i)
		    {	
		    	//Fill ranking list
		        ranklist[i].id = i; ranklist[i].fitness = pop[i].fx;

				/*******************Setting F and Cr************************/
				/*Generate F and Cr using a normal distribution with mean 
				taken randomly from storage*/
				r = rnd(0,H_maxsize-1); //take an index randomly  
				uF = M_F[r]; normal_distribution<double> Ndistribution_F(uF,0.1);
				uCr = M_Cr[r]; normal_distribution<double> Ndistribution_Cr(uCr,0.1);

		        evol_data[i].Cr = Ndistribution_Cr(rng); 
		        if (evol_data[i].Cr > 1.0) evol_data[i].Cr = 1.0; 
		        else if(evol_data[i].Cr < 0.0) evol_data[i].Cr = 0.0;

		        evol_data[i].F = Ndistribution_F(rng); 
		        if (evol_data[i].F > 1.0) evol_data[i].F = 1.0; 
		        while (evol_data[i].F <= 0.0) evol_data[i].F = Ndistribution_F(rng);

		        /*******************Setting p_best************************/
		        evol_data[i].p_best = rnd(0,int(rndreal(p_min,p_max)*NP)); //take an index within best range

		        /*******************Choosing a and b************************/
		        // randomly pick 2 different members
		        do evol_data[i].a = rnd(0,NP-1); while(evol_data[i].a==i); // from pop
		        do evol_data[i].b = rnd(0,memory.size()-1); while(evol_data[i].b==i || evol_data[i].b==evol_data[i].a); // from pop U archive

		        /*******************Get j_rand************************/
		        evol_data[i].j_rand = rnd(0,D-1);
		    }
		    //Rank population by fitness
		    sort(ranklist);//sort by fitness min => ranklist[0].fitness 

			//Load generated data and current population to device 
			hipMemcpy(evol_data_D,evol_data,NP*sizeof(evol_data_struct),hipMemcpyDefault);
			hipMemcpy(pop_D,pop,NP*sizeof(ind),hipMemcpyDefault);
			hipMemcpy(rank_D,ranklist,NP*sizeof(rank_ind),hipMemcpyDefault);
			hipMemcpy(memory_D,memory.data(),memory.size()*sizeof(ind),hipMemcpyDefault);

			//Lauch kernel: mutation,recombination and function evaluation 
			shade_engine<<<N_blocks,N_threads>>>(state_D,evol_data_D,pop_D,rank_D,memory_D,child_D,NP);
			F_D<<<N_blocks,N_threads>>>(Ovector_D,mem_D,Pvector_D,r25_D,r50_D,r100_D,s_D,w_D,OvectorVec_D,child_D,NP);
			hipMemcpy(child,child_D,NP*sizeof(ind),hipMemcpyDefault);
			
			//Selection
			for (i = 0; i < NP; ++i)
			{
				//Update FEs counter 
				glshade_current_FEs += 1; counter += 1;
		        if (child[i].fx <= pop[i].fx) // if better than target vector then:
		        {	
		        	//if strictly better then:
		        	if (child[i].fx < pop[i].fx)
		        	{
		        		A_tmp.push_back(pop[i]);//add defeated parent to external archive
		        		S_F[S_size] = evol_data[i].F;//record F
		        		S_Cr[S_size] = evol_data[i].Cr;//record Cr
		        		W[S_size] = pop[i].fx - child[i].fx;//record improvement
		        		S_size++;//increase storage size counter
		        	}

		        	//update global_best if needed 
		        	if (child[i].fx<global_best.fx && glshade_current_FEs<=glshade_stop_criterion)
		        	{
		        		global_best = child[i];
		        		global_best.FEs_when_found = glshade_current_FEs;
		        	}
		        	//Advance child to next generation
		        	pop[i] = child[i];
		        }
				if(glshade_current_FEs==1.2e5 || glshade_current_FEs==3e5 || glshade_current_FEs==6e5 || 
				glshade_current_FEs==9e5 || glshade_current_FEs==1.2e6 || glshade_current_FEs==1.5e6
				|| glshade_current_FEs==1.8e6 || glshade_current_FEs==2.1e6 || glshade_current_FEs==2.4e6 || 
				glshade_current_FEs==2.7e6 || glshade_current_FEs==3e6)
					fprintf(file_results,"%d,%d,%.2f,%.6e\n",glshade_current_FEs,ID,Rseed,global_best.fx);

			}
			//If F and Cr storages are non-empty
			if (S_size > 0)
			{	//Load F, Cr and W data to device  
				hipMemcpy(S_F_D,S_F,S_size*sizeof(double),hipMemcpyDefault);
				hipMemcpy(S_Cr_D,S_Cr,S_size*sizeof(double),hipMemcpyDefault);
				hipMemcpy(W_D,W,S_size*sizeof(double),hipMemcpyDefault);
				mean_WAWL<<<2,64>>>(S_Cr_D,S_F_D,W_D,S_size,mean_D); //Compute mean WA and mean WL in device
			}

			// Concurrently update best solution index
			update_best();

			// Concurrently check external archive
			A.insert(A.end(), A_tmp.begin(), A_tmp.end()); // add defeated parents to A
			apply_A_maintenance();//|A| must be less than or equal to popsize

			// Update M_CR and M_F
			if (S_size > 0)
			{	//Record means 
				hipMemcpy(mean,mean_D,2*sizeof(double),hipMemcpyDefault);
				M_F[k] = mean[1]; //weighted Lehmer mean (WL)
				M_Cr[k] = mean[0]; //weighted arithmetic mean (WA)
				k = (k + 1) % H_maxsize;
			}

			// Reset and go again
			S_size = 0;
			A_tmp.clear();
			memory.clear();
		}
	    //Free memory
	    hipFree(evol_data_D);
	    hipFree(pop_D);
	    hipFree(rank_D);
	    hipFree(memory_D);
	    hipFree(child_D);
	    hipFree(S_F_D);
	    hipFree(S_Cr_D);
	    hipFree(W_D);
	    hipFree(mean_D);
	}

	void free_memory()
	{
		free(pop);
		free(ranklist);
	    free(evol_data);
	    free(child);
	    free(S_F);
	    free(S_Cr);
	    free(W);
		free(M_F);
		free(M_Cr);
	}

};

